/*
 * Parallel bitonic sort using CUDA.
 * Compile with
 * nvcc -arch=sm_11 bitonic_sort.cu
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 128 // 2^8
#define BLOCKS 8192 // 2^14
#define NUM_VALS THREADS *BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
    double elapsed = ((double)(stop - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %.3fs\n", elapsed);
}

float random_float()
{
    return (float)rand() / (float)RAND_MAX;
}

void array_print(float *arr, int length)
{
    int i;
    for (i = 0; i < length; ++i)
    {
        printf("%1.3f ", arr[i]);
    }
    printf("\n");
}

void array_fill(float *arr, int length)
{
    srand(time(NULL));
    int i;
    for (i = 0; i < length; ++i)
    {
        arr[i] = random_float();
    }
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj) > i)
    {
        if ((i & k) == 0)
        {
            /* Sort ascending */
            if (dev_values[i] > dev_values[ixj])
            {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
        if ((i & k) != 0)
        {
            /* Sort descending */
            if (dev_values[i] < dev_values[ixj])
            {
                /* exchange(i,ixj); */
                float temp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = temp;
            }
        }
    }
}

/**
 * Inplace bitonic sort using CUDA.
 */
void bitonic_sort(float *values)
{
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);   /* Number of blocks   */
    dim3 threads(THREADS, 1); /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= NUM_VALS; k <<= 1)
    {
        /* Minor step */
        for (j = k >> 1; j > 0; j = j >> 1)
        {
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }
    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

void writefile(char *filename, float *buffer, int num)
{
    FILE *fp;
    fp = fopen(filename, "w");
    for (int j = 0; j < num; j++)
    {
        fprintf(fp, "%0.0f\n", *(buffer + j));
    }
    fclose(fp);
}

int main(int argc, char *argv[])
{
    clock_t start, stop;

    if (argc != 3)
    {
        printf("Invalid argument count.  %s accepts 1-4 arguments, %d given\n",
               argv[0], argc);
        return -1;
    }

    float *values = (float *)malloc(NUM_VALS * sizeof(float));
    // array_fill(values, NUM_VALS);
    if (values == NULL)
    {
        printf("Insufficient host memory to allocate at %d", __LINE__);
        return -3;
    }

    start = clock();
    FILE *fin = fopen(argv[1], "r");
    for (int i = 0; i < NUM_VALS; i++)
    {
        if (EOF == fscanf(fin, "%f ", &values[i]))
        {
            break;
        }
    }

    bitonic_sort(values); /* Inplace */

    writefile(argv[2], values, NUM_VALS);

    stop = clock();
    print_elapsed(start, stop);
    free(values);
}