#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>

/**
 * mergesort.cu
 * a one-file c++ / cuda program for performing mergesort on the GPU
 * While the program execution is fairly slow, most of its runnning time
 *  is spent allocating memory on the GPU.
 * For a more complex program that performs many calculations,
 *  running on the GPU may provide a significant boost in performance
 */

// helper for main()
long readList(long **);

// data[], size, threads, blocks,
void mergesort(long *, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long *, long *, long, long, long, dim3 *, dim3 *);
__device__ void gpu_bottomUpMerge(long *, long *, long, long, long);

// profiling
int tm();

#define min(a, b) (a < b ? a : b)
#define MAX_LENGTH 1000000001

int readfile(char *filename, long *buffer)
{
    FILE *fh;
    int i = 0;
    fh = fopen(filename, "r");
    if (fh != NULL)
    {
        char line[20]; /* or other suitable maximum line size */
        while (fgets(line, sizeof line, fh) != NULL)
        { /* read a line */
            *(buffer + i++) = strtol(line, 0, 10);
        }
        fclose(fh);
    }
    return i;
}

void writefile(char *filename, long *buffer, int num)
{
    FILE *fp;
    fp = fopen(filename, "w");
    for (int j = 0; j < num; j++)
    {
        fprintf(fp, "%ld\n", *(buffer + j));
    }
    fclose(fp);
}

void printHelp(char *program)
{

    std::cout
        << "usage: " << program << " [-xyzXYZvio]\n"
        << '\n'
        << "-x, -y, -z are numbers of threads in each dimension. On my machine\n"
        << "  the correct number is x*y*z = 32\n"
        << '\n'
        << "-X, -Y, -Z are numbers of blocks to use in each dimension. Each block\n"
        << "  holds x*y*z threads, so the total number of threads is:\n"
        << "  x*y*z*X*Y*Z\n"
        << '\n'
        << "-v prints out extra info\n"
        << '\n'
        << "-i input file dir\n"
        << '\n'
        << "-o output file dir\n"
        << '\n'
        << "? prints this message and exits\n"
        << '\n'
        << "example: ./mergesort -x 8 -Y 10 -v\n"
        << '\n'
        << "Reads in a list of integer numbers from stdin, and performs\n"
        << "a bottom-up merge sort:\n"
        << '\n'
        << "Input:          8 3 1 9 1 2 7 5 9 3 6 4 2 0 2 5\n"
        << "Threads: |    t1    |    t2    |    t3    |    t4    |\n"
        << "         | 8 3 1 9  | 1 2 7 5  | 9 3 6 4  | 2 0 2 5  |\n"
        << "         |  38 19   |  12 57   |  39 46   |  02 25   |\n"
        << "         |   1398   |   1257   |   3469   |   0225   |\n"
        << "         +----------+----------+----------+----------+\n"
        << "         |          t1         |          t2         |\n"
        << "         |       11235789      |       02234569      |\n"
        << "         +---------------------+---------------------+\n"
        << "         |                     t1                    |\n"
        << "         |      0 1 1 2 2 2 3 3 4 5 5 6 7 8 9 9      |\n"
        << '\n'
        << '\n';
}

bool verbose;
int main(int argc, char **argv)
{
    char *inputFile;
    char *outputFile;
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    //
    // Parse argv
    //
    tm();
    for (int i = 1; i < argc; i++)
    {
        if (argv[i][0] == '-' && argv[i][1] && !argv[i][2])
        {
            char arg = argv[i][1];
            unsigned int *toSet = 0;
            switch (arg)
            {
            case 'x':
                toSet = &threadsPerBlock.x;
                break;
            case 'y':
                toSet = &threadsPerBlock.y;
                break;
            case 'z':
                toSet = &threadsPerBlock.z;
                break;
            case 'X':
                toSet = &blocksPerGrid.x;
                break;
            case 'Y':
                toSet = &blocksPerGrid.y;
                break;
            case 'Z':
                toSet = &blocksPerGrid.z;
                break;
            case 'v':
                verbose = true;
                break;
            case 'i':
                inputFile = argv[++i];
                break;
            case 'o':
                outputFile = argv[++i];
                break;
            default:
                std::cout << "unknown argument: " << arg << '\n';
                printHelp(argv[0]);
                return -1;
            }

            if (toSet)
            {
                i++;
                *toSet = (unsigned int)strtol(argv[i], 0, 10);
            }
        }
        else
        {
            if (argv[i][0] == '?' && !argv[i][1])
                std::cout << "help:\n";
            else
                std::cout << "invalid argument: " << argv[i] << '\n';
            printHelp(argv[0]);
            return -1;
        }
    }

    if (verbose)
    {
        std::cout << "parse argv " << tm() << " microseconds\n";
        std::cout << "\nthreadsPerBlock:"
                  << "\n  x: " << threadsPerBlock.x
                  << "\n  y: " << threadsPerBlock.y
                  << "\n  z: " << threadsPerBlock.z
                  << "\n\nblocksPerGrid:"
                  << "\n  x:" << blocksPerGrid.x
                  << "\n  y:" << blocksPerGrid.y
                  << "\n  z:" << blocksPerGrid.z
                  << "\n\n total threads: "
                  << threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                         blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z
                  << "\n\n";
    }

    //
    // Read numbers from stdin
    //
    long *data = (long *)malloc(MAX_LENGTH * sizeof(long));
    if (verbose)
        std::cout << "reading input file" << inputFile << "\n";
    long size = readfile(inputFile, data);
    // long size = readList(&data);

    if (!size)
        return -1;

    if (verbose)
        std::cout << "sorting " << size << " numbers\n\n";

    // merge-sort the data
    mergesort(data, size, threadsPerBlock, blocksPerGrid);

    writefile(outputFile, data, int(size));

    tm();

    if (verbose)
    {
        //
        // Print out the list
        //
        for (int i = 0; i < size; i++)
        {
            std::cout << data[i] << '\n';
        }
        std::cout << "print list to stdout: " << tm() << " microseconds\n";
    }
}

void mergesort(long *data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid)
{

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long *D_data;
    long *D_swp;
    dim3 *D_threads;
    dim3 *D_blocks;

    // Actually allocate the two arrays
    tm();
    checkCudaErrors(hipMalloc((void **)&D_data, size * sizeof(long)));
    checkCudaErrors(hipMalloc((void **)&D_swp, size * sizeof(long)));
    if (verbose)
        std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    checkCudaErrors(hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice));
    if (verbose)
        std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";

    //
    // Copy the thread / block info to the GPU as well
    //
    checkCudaErrors(hipMalloc((void **)&D_threads, sizeof(dim3)));
    checkCudaErrors(hipMalloc((void **)&D_blocks, sizeof(dim3)));

    if (verbose)
        std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    checkCudaErrors(hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

    if (verbose)
        std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    long *A = D_data;
    long *B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1)
    {
        long slices = size / ((nThreads)*width) + 1;

        if (verbose)
        {
            std::cout << "mergeSort - width: " << width
                      << ", slices: " << slices
                      << ", nThreads: " << nThreads << '\n';
            tm();
        }

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        if (verbose)
            std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    tm();
    checkCudaErrors(hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost));
    if (verbose)
        std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";

    // Free the GPU memory
    checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
    if (verbose)
        std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3 *threads, dim3 *blocks)
{
    int x;
    return threadIdx.x +
           threadIdx.y * (x = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x * (x *= threads->z) +
           blockIdx.y * (x *= blocks->z) +
           blockIdx.z * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long *source, long *dest, long size, long width, long slices, dim3 *threads, dim3 *blocks)
{
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices,
         middle,
         end;

    for (long slice = 0; slice < slices; slice++)
    {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long *source, long *dest, long start, long middle, long end)
{
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++)
    {
        if (i < middle && (j >= end || source[i] < source[j]))
        {
            dest[k] = source[i];
            i++;
        }
        else
        {
            dest[k] = source[j];
            j++;
        }
    }
}

// read data into a minimal linked list
typedef struct
{
    int v;
    void *next;
} LinkNode;

// helper function for reading numbers from stdin
// it's 'optimized' not to check validity of the characters it reads in..
long readList(long **list)
{
    tm();
    long v, size = 0;
    LinkNode *node = 0;
    LinkNode *first = 0;
    while (std::cin >> v)
    {
        LinkNode *next = new LinkNode();
        next->v = v;
        if (node)
            node->next = next;
        else
            first = next;
        node = next;
        size++;
    }

    if (size)
    {
        *list = new long[size];
        LinkNode *node = first;
        long i = 0;
        while (node)
        {
            (*list)[i++] = node->v;
            node = (LinkNode *)node->next;
        }
    }

    if (verbose)
        std::cout << "read stdin: " << tm() << " microseconds\n";

    return size;
}

//
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm()
{
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}
