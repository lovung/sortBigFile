#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "common/io.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include "common/cuPrintf.cu"

/*********************** Data Definitions ********************************/
#define THREADS_PER_BLOCK 256

//These Inline Functions are used in the CPU Quicksort Implementation
#define swap(A, B)      \
    {                   \
        float temp = A; \
        A = B;          \
        B = temp;       \
    }
//#define compswap(A,B) if(B < A) swap(A,B)

//These Data Structs are used in the GPU Quicksort Implementation

typedef struct vars
{
    int l;
    int r;
    int leq;
} vars;

/*********************** CPU QUICKSORT IMPLEMENTATION ***********************/

/* csort
 *
 * This function is an implementation of 'Quicksort with three-way 
 * partitioning' from 'Algorithms in C' (Program 7.5, page 326).
 *
 * Parameters:
 * ls: The list of floating points being sorted
 * l: index of the left most item in ls being sorted at the moment
 * r: index of the right most item in ls being sorted at the moment
 */
void csort(float ls[], int l, int r)
{
    int i, j, k, p, q;
    float v;
    if (r <= l)
        return;
    v = ls[r];
    i = l - 1;
    j = r;
    p = l - 1;
    q = r;
    for (;;)
    {
        while (ls[++i] < v)
            ;
        while (v < ls[--j])
            if (j == l)
                break;
        if (i >= j)
            break;
        swap(ls[i], ls[j]);
        if (ls[i] == v)
        {
            p++;
            swap(ls[p], ls[i]);
        }
        if (v == ls[j])
        {
            q--;
            swap(ls[q], ls[j]);
        }
    }
    swap(ls[i], ls[r]);
    j = i - 1;
    i++;
    for (k = l; k < p; k++, j--)
        swap(ls[k], ls[j]);
    for (k = r - 1; k > q; k--, i++)
        swap(ls[k], ls[i]);

    csort(ls, l, j);
    csort(ls, i, r);
}

/* cpu_quicksort
 *
 * This function is called to sort the floating point array using a CPU-based
 * implementation of quicksort. Its purpose is to set up the timing functions
 * to wrap the recursive 'csort' function which does the actual sorting
 *
 * Parameters:
 * unsorted: The array of floating point numbers to be sorted
 * length: the length of the unsorted & sorted arrays
 * sorted: an output parameter, will store the final, sorted array.
 *
 * Output:
 * time: This function should return the amount of time taken to sort the list.
 */
double cpu_quicksort(float unsorted[], int length, float sorted[])
{

    for (int i = 0; i < length; i++)
        sorted[i] = unsorted[i];

    clock_t start, end;
    double time;
    start = clock();
    csort(sorted, 0, length - 1);
    end = clock();
    time = ((double)end - start) / CLOCKS_PER_SEC;

    return time;
}

/***************************** GPU IMPLEMENTATION ****************************/

/* gpuPartitionSwap
 *
 * This kernel function is called recursively by the host. Its purpose is to, 
 * given a pivot value, partition and swap items in the section of the input
 * array bounded by the l & r indices, then store the pivot in the correct
 * location.
 *
 * Parameters:
 * input: The unsorted (or partially sorted) input data
 * output: The aptly named output parameter, it is the same as input, but all
 *         floating points within (l,r) have been partitioned and swapped.
 * endpts: This is a custom data struct meant to 
 *         a) hold a counter variable in global memory
 *         b) pass the l' and r' parameters back to the host to the left and
 *            right of the positioned pivot item.
 * pivot: This is the pivot value, about which all items in (l,r) are being
 *        swapped.
 * l: the left index bound on input & output
 * r: the right index bound on input & output
 * d_leq: an array of offset values, storedin global device memory
 * nBlocks: The total number of blocks, to be used to determine the location
 *          of insertion of the pivot.
 *
 */
__global__ void gpuPartitionSwap(float *input, float *output, vars *endpts,
                                 float pivot, int l, int r,
                                 int d_leq[],
                                 int d_gt[], int *d_leq_val, int *d_gt_val,
                                 int nBlocks)
{
    //copy a section of the input into shared memory
    __shared__ float bInput[THREADS_PER_BLOCK];
    __syncthreads();
    int idx = l + blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    __shared__ int lThisBlock, rThisBlock;
    __shared__ int lOffset, rOffset;

    if (threadIdx.x == 0)
    {
        d_leq[blockIdx.x] = 0;
        d_gt[blockIdx.x] = 0;
        *d_leq_val = 0;
        *d_gt_val = 0;
        //cuPrintf("STARTING NEXT SET!  l: %d, r: %d\n", l, r);
    }
    __syncthreads();

    if (idx <= (r - 1))
    {
        bInput[threadIdx.x] = input[idx];
        //cuPrintf("read %f at input [%d]\n", bInput[threadIdx.x], idx);

        //make comparison against the pivot, setting 'status' and updating the counter (if necessary)
        if (bInput[threadIdx.x] <= pivot)
        {
            //cuPrintf("input:  %f < pivot: %f\n", bInput[threadIdx.x], pivot);
            atomicAdd(&(d_leq[blockIdx.x]), 1);
        }
        else
        {
            //cuPrintf("input:  %f > pivot: %f\n", bInput[threadIdx.x], pivot);
            atomicAdd(&(d_gt[blockIdx.x]), 1);
        }
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        lThisBlock = d_leq[blockIdx.x];
        lOffset = l + atomicAdd(d_leq_val, lThisBlock);
    }
    if (threadIdx.x == 1)
    {
        rThisBlock = d_gt[blockIdx.x];
        rOffset = r - atomicAdd(d_gt_val, rThisBlock);
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        //cuPrintf("lThisBlock, rThisBlock: %d, %d\n", lThisBlock, rThisBlock);
        //cuPrintf("lOffset: %d, rOffset: %d\n", lOffset, rOffset);
        /*
    for(int i = 1; i <= blockIdx.x; i++){
      lOffset += d_leq[i - 1];
      rOffset -= d_gt[i - 1];
      }
*/

        int m = 0;
        int n = 0;
        for (int j = 0; j < THREADS_PER_BLOCK; j++)
        {
            int chk = l + blockIdx.x * THREADS_PER_BLOCK + j;
            if (chk <= (r - 1))
            {
                if (bInput[j] <= pivot)
                {
                    //cuPrintf("writing testval %f to loutput[%d = %d + %d]\n",
                    //     bInput[j], lOffset+m, lOffset, m);
                    output[lOffset + m] = bInput[j];
                    ++m;
                }
                else
                {
                    //cuPrintf("writing testval %f to routput[%d = %d - %d]\n",
                    //      bInput[j], rOffset-n, rOffset, n);
                    output[rOffset - n] = bInput[j];
                    ++n;
                }
            }
        }
    }

    __syncthreads();

    if ((blockIdx.x == 0) && (threadIdx.x == 0))
    {
        int pOffset = l;
        for (int k = 0; k < nBlocks; k++)
            pOffset += d_leq[k];

        output[pOffset] = pivot;
        //cuPrintf("writing pivot %f to output[%d]\n", pivot, pOffset);
        endpts->l = (pOffset - 1);
        endpts->r = (pOffset + 1);
    }

    return;
}

void gqSort(float ls[], int l, int r, int length)
{
    //if (r - l) >= 1
    if ((r - l) >= 1)
    {

        //0.  print list
        /*    printf("List is: ");
    for (int temp = 0; temp < length; temp++){
      printf("%f ", ls[temp]);
    }
    printf("\n");    
*/

        //1. grab pivot
        float pivot = ls[r];

        //2. set-up gpu vars
        int numBlocks = (r - l) / THREADS_PER_BLOCK;
        if ((numBlocks * THREADS_PER_BLOCK) < (r - l))
            numBlocks++;

        float *d_ls;
        float *d_ls2;
        vars endpts;
        endpts.l = l;
        endpts.r = r;

        vars *d_endpts;
        int *d_leq, *d_gt, *d_leq_val, *d_gt_val;
        int size = sizeof(float);
        hipMalloc(&(d_ls), size * length);
        hipMalloc(&(d_ls2), size * length);
        hipMalloc(&(d_endpts), sizeof(vars));
        hipMalloc(&(d_leq), 4 * numBlocks);
        hipMalloc(&(d_gt), 4 * numBlocks);
        hipMalloc(&d_leq_val, 4);
        hipMalloc(&d_gt_val, 4);
        hipMemcpy(d_ls, ls, size * length, hipMemcpyHostToDevice);
        hipMemcpy(d_ls2, ls, size * length, hipMemcpyHostToDevice);

        //3. call gpuPartition function
        gpuPartitionSwap<<<numBlocks, THREADS_PER_BLOCK>>>(d_ls, d_ls2, d_endpts, pivot, l, r, d_leq, d_gt, d_leq_val, d_gt_val, numBlocks);

        //4. Retrieve sorted list and other variables
        hipMemcpy(ls, d_ls2, size * length, hipMemcpyDeviceToHost);
        hipMemcpy(&(endpts), d_endpts, sizeof(vars), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();
        //cudaPrintfDisplay(stdout,true);
        //5.recursively call on left/right sections of list generated by gpuPartition

        hipFree(d_ls);
        hipFree(d_ls2);
        hipFree(d_endpts);
        hipFree(d_leq);
        hipFree(d_gt);

        if (endpts.l >= l)
            gqSort(ls, l, endpts.l, length);
        if (endpts.r <= r)
            gqSort(ls, endpts.r, r, length);
    }

    return;
}

/* gpu_quicksort
 *
 * This is a function meant to set up the custom 'data' struct array
 * used by the gpu implementation of quicksort, as well as to calculate
 * the time of execution of the sorting algorithm.
 *
 * Parameters:
 * unsorted: The array of floats to be sorted
 * length: The length of the unsorted and sorted arrays
 * sorted: An output parameter, to be filled with the sorted array.
 *
 * Output:
 * time: This function returns the time of execution required by the
 *       sorting algorithm
 */
double gpu_quicksort(float unsorted[], int length, float sorted[])
{
    time_t start, end;
    double time;
    for (int i = 0; i < length; i++)
        sorted[i] = unsorted[i];

    start = clock();
    gqSort(sorted, 0, length - 1, length);
    end = clock();
    time = ((double)end - start) / CLOCKS_PER_SEC;
    return time;
}

/* quicksort
 * 
 * This function is called by main to populate a result, testing the CPU
 * and GPU implementations of quicksort.
 *
 * Parameters:
 * unsorted: an unsorted array of floating points
 * length: the length of the unsorted array
 * result: an output parameter to be filled with the results of the cpu and gpu
 *         implementations of quicksort.
 *
 */
void quicksort(float unsorted[], int length, Result *result)
{
    result = (Result *)malloc(sizeof(Result));

    cudaPrintfInit();

    if (result == NULL)
    {
        fprintf(stderr, "Out of Memory\n");
        exit(1);
    }
    strcpy(result->tname, "Quick Sort");
    float sorted[2][length];

    result->cpu_time = cpu_quicksort(unsorted, length, sorted[0]);
    result->gpu_time = gpu_quicksort(unsorted, length, sorted[1]);

    //check that sorted[0] = sorted[1];
    int n = 0;
    for (int i = 0; i < length; i++)
    {
        if (sorted[0][i] != sorted[1][i])
            n++;
        //      printf("CPU #%d: %f\t", i, sorted[0][i]);
        //printf("GPU #%d: %f", i, sorted[1][i]);
        //printf("\n", i, sorted[0][i]);
    }

    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    if (n != 0)
    {
        fprintf(stdout, "There were %d discrepencies between the CPU and GPU QuickSort algorithms\n", n);
    }
    printf("CPU Time: %.10f, GPU time: %.10f\n", result->cpu_time, result->gpu_time);
    return;
}

void readfile(char *filename, float *buffer, int num)
{
    FILE *fh;
    int i = 0;
    fh = fopen(filename, "r");
    if (fh != NULL)
    {
        char line[20]; /* or other suitable maximum line size */
        while (fgets(line, sizeof line, fh) != NULL)
        { /* read a line */
            *(buffer + i++) = atoi(line);
        }
        fclose(fh);
    }
}

void writefile(char *filename, float *buffer, int num)
{
    FILE *fp;
    fp = fopen(filename, "w");
    for (int j = 0; j < num; j++)
    {
        fprintf(fp, "%0.0f\n", *(buffer + j));
    }
    fclose(fp);
}

int main(int argc, char *argv[])
{
    int len;
    float *h_list;
    float *sorted;

    double time;
    cudaPrintfInit();

    if ((argc > 4) || argc < 2)
    {
        printf("Invalid argument count.  %s accepts 1-4 arguments, %d given\n",
               argv[0], argc);
        return -1;
    }

    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);

    FILE *fin = fopen(argv[1], "r");

    if (fin == NULL)
    {
        printf("Could not open file: %s", argv[1]);
        return -2;
    }

    len = atoi(argv[3]);
    printf("Len: %d\n", len);

    h_list = (float *)malloc(len * sizeof(float));
    if (h_list == NULL)
    {
        printf("Insufficient host memory to allocate at %d", __LINE__);
        return -3;
    }

    for (int i = 0; i < len; i++)
    {
        if (EOF == fscanf(fin, "%f ", &h_list[i]))
        {
            break;
        }
    }

    sorted = new float[len];
    // printf("START SORTING!\n");
    time = gpu_quicksort(h_list, len, sorted);
    // printf("FINISH SORTING!\n");
    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    writefile(argv[2], sorted, len);
    printf("Time: %f\n", time);

    return 0;
}
